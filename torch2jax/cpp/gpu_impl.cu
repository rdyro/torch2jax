#include "gpu_impl.h"

py::dict GPURegistrations() {
  py::dict dict;
  // dict["gpu_torch_call_f32"] =
  // encapsulateFunction(gpu_apply_torch_call<float>);
  // dict["gpu_torch_call_f64"] =
  // encapsulateFunction(gpu_apply_torch_call<double>);
  dict["gpu_torch_call"] = encapsulateFunction(gpu_apply_torch_call);
  return dict;
}

// template <typename T>
void gpu_apply_torch_call(hipStream_t stream, void **buffers,
                          const char *opaque, size_t opaque_len) {
  /* ---------------------------------------------------------------------------
  The GPU version of this routine just deserializes the descriptor and calls the
  main `apply_torch_call` routine.
  --------------------------------------------------------------------------- */

  DescriptorDataAccessor da(reinterpret_cast<const int64_t*>(opaque), nullptr);
  DynamicTorchCallDescriptor d;
  deserialize_descriptor(d, da);

  // apply_torch_call<T>(buffers, d);
  apply_torch_call(buffers, d);
}

TorchCallDevice actual_cuda_device(const TorchCallDevice& device_desc, void* buffer) {
#ifdef TORCH2JAX_WITH_CUDA
    hipDevice_t device_ordinal;
    hipError_t err = hipPointerGetAttribute((void*)&device_ordinal, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL, (hipDeviceptr_t)buffer);
    if (err != hipSuccess) return device_desc;
    TorchCallDevice new_device_desc = device_desc;
    new_device_desc.index = device_ordinal;
    return new_device_desc;
#else
  return device_desc;
#endif
}
